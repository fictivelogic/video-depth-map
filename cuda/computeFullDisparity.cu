#include "hip/hip_runtime.h"
#include <math.h>

// #define IMAGE_HEIGHT 260
// #define img_width 360

__global__ void

computeFullDisparity(const float * pixelsFullR,
                     const float * pixelsFullL,
                     float * dHalf,
                     float * disparityFull
                     const int img_width,
                     const int img_height,
                     const bool * foregroundFullR,
                     const bool * foregroundFullL)
{
    int ind_i = 8 * threadIdx.x + 64 * blockIdx.x;
    int ind_j = threadIdx.y + 64 * blockIdx.y;
    const int end_i = ind_i + 8;

    for (; i < end_i; i++)
    {
        if (foregroundFullL[i]==1)
        {
            float * matchingCost = new float [5];
            int k = ((i/img_width)/2) + ((i%img_width)/2);
            int prevdisp = (int) floorf(dHalf[k]);
            int initial_j = (2* prevdisp) -2;
            int max_j = 0;
            for (int j = initial_j; j <= initial_j + 4; j++)
            {
                if ( (i % img_width != 0) && ((j+i) % img_width  == 0) )
                {
                    break;
                }
                else
                {
                    if (foregroundFullR[i+j] == 1)
                    {
                        matchingCost[j - initial_j] = 0;
                        for (int m = 0; m < 3; m++)
                        {
                            matchingCost[j - initial_j] = powf((pixelsQuartL[(3*i) + m] - pixelsQuartR[(3*(i+j)) +m]),2);
                        }
                        matchingCost[j - initial_j] /= 3;
                        max_j = j - initial_j;
                    }
                    else
                    {
                        matchingCost[j - initial_j] = 1000;
                    }
                }
            }
            float curr_min = 5000;
            for (int l = 0; l <= max_j; l++)
            {
                if (curr_min > matchingCost[l])
                {
                    curr_min = matchingCost[l];
                }
            }
            disparityFull[i] = sqrt(curr_min);
            delete [] matchingCost;
        }
        else
        {
            disparityFull[i] = 20;
        }
    }
    //copy to CPU mem
}