#include "hip/hip_runtime.h"
__global__ void


computeDisparity(const Color * pixelsFullR,
                 const Color * pixelsHalfR,
                 const Color * pixelsQuartR,
                 const Color * pixelsFullL,
                 const Color * pixelsHalfL,
                 const Color * pixelsQuartL,
                 const int h,
                 const int v,
                 const int sFull,
                 const bool * foregroundFullR,
                 const bool * foregroundHalfR,
                 const bool * foregroundHQuarterR,
                 const bool * foregroundFullL,
                 const bool * foregroundHalfL,
                 const bool * foregroundHQuarterL,
                 float * disparityFull)
{
    //Determine indexes based on Thread ID's

    //Initial disparity measure calculated differently
    int sQuart = sFull / 4;
    float dQuarter[] = new float [h * v * 0.0625]; //Assuming h and v are divisible by 4
    
    for (int i = 0; i < h*v; i++)
    {
        if foregroundHQuarterR[i] == 1
        {
            for (int j = 0; j < sQuart; j++)
            {
                if (j+i >= (i+1)*h)
                    break
                else
                {
                    if (foregroundHQuarterL[i+j]==1)
                    {
                        matchingCost[j] = //compute matching cost from openCV
                    }
                    else
                    {
                        matchingCost[j] = inf; //Syntax?
                    }
                }
            }
            dQuarter[i] = min(matchingCost);
        }
        else
            dQuarter[i] = 20; //Decide on what to do for this case
    }

}
