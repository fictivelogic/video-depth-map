#include "hip/hip_runtime.h"
#include <math.h>
#include "computeFullDisparity.cu"

#define IMAGE_HEIGHT 260
#define IMAGE_WIDTH 360

__global__ void

computeHalfDisparity(const float * pixelsFullR,
                     const float * pixelsHalfR,
                     const float * pixelsFullL,
                     const float * pixelsHalfL,
                     float * dQuarter,
                     float * dHalf,
                     float * disparityFull,
                     const bool * foregroundFullR,
                     const bool * foregroundHalfR,
                     const bool * foregroundFullL,
                     const bool * foregroundHalfL)
{

    for (int i = 0; i < IMAGE_WIDTH * IMAGE_HEIGHT/4; i++)
    {
        if (foregroundHalfL[i]==1)
        {
            float * matchingCost = new float [5];
            int k = ((i/(IMAGE_WIDTH/2) )/2) + ((i%(IMAGE_WIDTH/2) )/2);
            int prevdisp = (int) floorf(dQuarter[k]);
            int initial_j = (2* prevdisp) -2;
            int max_j = 0;
            for (int j = initial_j; j <= initial_j + 4; j++)
            {
                if ( (i % (IMAGE_WIDTH/2) != 0) && ((j+i) % (IMAGE_WIDTH/4)  == 0) )
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[i+j] == 1)
                    {
                        matchingCost[j - initial_j] = 0;
                        for (int m = 0; m < 3; m++)
                        {
                            matchingCost[j - initial_j] = square(pixelsQuartL[(3 * i) + m] - pixelsQuartR[(3*(i+j)) + m]);
                        }
                        matchingCost[j - initial_j] /= 3;
                        max_j = j - initial_j;
                    }
                    else
                    {
                        matchingCost[j - initial_j] = 1000;
                    }
                }
            }
            float curr_min = 5000;
            for (int l = 0; l <= max_j; l++)
            {
                if (curr_min > matchingCost[l])
                {
                    curr_min = matchingCost[l];
                }
            }
            dHalf[i] = sqrt(curr_min);
            delete [] matchingCost;
        }
        else
        {
            dHalf[i] = 20;
        }
    }

    computeFullDisparity(const float * pixelsFullR,
                     const float * pixelsFullL,
                     float * dHalf,
                     float * disparityFull,
                     const bool * foregroundFullR,
                     const bool * foregroundFullL);

}