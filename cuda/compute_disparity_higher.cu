#include "hip/hip_runtime.h"
// Simplified attempt
#define STARTING_MATCHING_COST 100.0f
struct pixel {
    float R;
    float G;
    float B;
};

__global__ void computeDisparity(
    const struct pixel * imageR, 
    const struct pixel * imageL,
    const int image_height,
    const int image_width,
    const char * foregroundR,
    const char * foregroundL,
    float * disparity_input,
    float * disparity_output)
{
    // Local variables:
    long int pixel_index = blockIdx.x * blockDim.x + threadIdx.x; // Index of current pixel
    float matching_cost = 0.0;
    float min_matching_cost = STARTING_MATCHING_COST;
    long int min_cost_offset = 0; 
    long int offset_pixel_index = 0;

    while(pixel_index < image_height * image_width) { 
        // while... the thread index hasn't gone outside the image dimensions
        if (true) { //foregroundL[pixel_index] == 1) {
            // Calculate matching cost for this foreground pixel
            // ensure that we are not going over the end of the pixel row
            offset_pixel_index = pixel_index;
            min_matching_cost = STARTING_MATCHING_COST;
            min_cost_offset = 0;
            for (int offset = 0; offset < window_size; offset++) { 
                if ((pixel_index % image_width) + offset >= image_width) {
                    break;
                }
                matching_cost =  powf(imageL[pixel_index].R - imageR[offset_pixel_index].R, 2);
                matching_cost += powf(imageL[pixel_index].G - imageR[offset_pixel_index].G, 2);
                matching_cost += powf(imageL[pixel_index].B - imageR[offset_pixel_index].B, 2);
                if (matching_cost < min_matching_cost) {
                    min_matching_cost = matching_cost; 
                    min_cost_offset = offset;
                }
                offset_pixel_index++;
            }

            if (min_matching_cost == STARTING_MATCHING_COST) {
                disparity_output[pixel_index] = 0;
            } else {
                disparity_output[pixel_index] = powf(min_cost_offset, 2); //  + image_width;
            }
        } else {
            disparity_output[pixel_index] = -1;
        }
        pixel_index += blockDim.x * gridDim.x;
    }
}

