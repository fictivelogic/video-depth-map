#include "hip/hip_runtime.h"
__global__ void


// [260, 720, 3] 
// [260, 0-359, 3] would be LEFT [260, 360:719, 3] would be RIGHT

computeDisparity(const Color * pixelsFullR,
                 const Color * pixelsHalfR,
                 const Color * pixelsQuartR,
                 const Color * pixelsFullL,
                 const Color * pixelsHalfL,
                 const Color * pixelsQuartL,
                 const int h,
                 const int v,
                 const int sFull,
                 const bool * foregroundFullR,
                 const bool * foregroundHalfR,
                 const bool * foregroundHQuarterR,
                 const bool * foregroundFullL,
                 const bool * foregroundHalfL,
                 const bool * foregroundHQuarterL,
                 float * disparityFull)
{
    //Determine indexes based on Thread ID's

    //Initial disparity measure calculated differently
    int sQuart = sFull / 4.0;
    int dQuarter[] = new int [h * v * 0.0625]; //Assuming h and v are divisible by 4
    
    for (int i = 0; i < (h*v * 0.0625); i++)
    {
        if (foregroundHQuarterL[i] == 1)
        {
            int matchingCost[] = new int [sQuart];
            for (int j = i; j < i + sQuart; j++)
            {
                if (j+i >= (i+1)*h * 0.25) //Check if the index within windows is in pixel block row range
                {
                    break;
                }
                else
                {
                    if (foregroundHQuarterR[i+j]==1)
                    {
                        matchingCost[j-i] = 3;//compute matching cost from openCV
                    }
                    else
                    {
                        matchingCost[j-i] = 1000; //Syntax?
                    }
                }
            }
            dQuarter[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else
        {
            dQuarter[i] = 20; //Decide on what to do for this case
        }
    }

    int dHalf = new int [h*v* 0.25];
    for (int i = 0; i < (h * v * 0.25); i++)
    {
        if (foregroundHalfL[i] == 1)
        {
            int matchingCost[] = new int [4];
            int prevdisp = dQuarter[i/2]; //intentionally taking advantage of int division
            for (int j = 2 * prevdisp - 2; j < 2 * prevdisp + 2; j++)
            {
                if (j+i * h * 0.5)
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[j+i] == 1)
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 3; //compute matching cost
                    }
                    else
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 1000;
                    }
                }

            }
            dHalf[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else {
            dHalf[i] = 20;
        }
    }

    //int disparityFull = new int [h*v* 0.25];
    for (int i = 0; i < (h * v); i++)
    {
        if (foregroundFullL[i] == 1)
        {
            int matchingCost[] = new int [4];
            int prevdisp = dHalf[i/2]; //intentionally taking advantage of int division
            for (int j = 2 * prevdisp - 2; j < 2 * prevdisp + 2; j++)
            {
                if (j+i * h)
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[j+i] == 1)
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 3; //compute matching cost
                    }
                    else
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 1000;
                    }
                }

            }
            disparityFull[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else {
            disparityFull[i] = 20;
        }
    }

}

computeInitialDisparity(const float *** pixelsFullR,
                        const Color *** pixelsHalfR,
                        const Color *** pixelsQuartR,
                        const Color *** pixelsFullL,
                        const Color *** pixelsHalfL,
                        const Color *** pixelsQuartL,
                        const int sFull,
                        const bool ** foregroundFullR,
                        const bool ** foregroundHalfR,
                        const bool ** foregroundQuarterR,
                        const bool ** foregroundFullL,
                        const bool ** foregroundHalfL,
                        const bool ** foregroundQuarterL,
                        float * disparityFull)
{
    int sQuart = sFull / 4.0; //sFull is the max window we would want for full res pixel block searching
    float ** dQuarter = new int [h * v * 0.0625]; //Assuming h and v are divisible by 4
    
    for (int k = 0; k < IMAGE_HEIGHT/4; k++)
    {
        for (int i = 0; i < IMAGE_WIDTH/4; i++)
        {
            if (foregroundQuarterL[k][i] == 1)
            {
                int * matchingCost = new int [sQuart];
                int max_j = 0;
                for (int j = i; j < i + sQuart; j++)
                {
                    if (j+i >= (i+1)*h * 0.25) //Check if the index within windows is in pixel block row range
                    {
                        break;
                    }
                    else
                    {
                        if (foregroundQuarterR[i+j]==1)
                        {
                            matchingCost[j-i] = 0;
                            for (int m = 0; m < 3; m++)
                            {
                                matchingCost[j-i] += square(pixelsQuartL[k][i][m] - pixelsQuartR[k][i+j][m]);
                            }
                            matchingCost[j-i] /= 3;
                            float tempCost = sqrt(matchingCost[j-i]);
                            matchingCost[j-i] = tempCost;
                            max_j = j;
                        }
                        else
                        {
                            matchingCost[j-i] = 1000;// might have to also update max_j
                        }
                    }
                }
                float curr_min = 5000;
                for (int l = 0; l <= max_j; l++)
                {
                    if (curr_min > matchingCost[l])
                    {
                        curr_min = matchingCost[l];
                    }
                }
                dQuarter[k][i] = curr_min;
                delete [] matchingCost;
            }
            else
            {
                dQuarter[k][i] = 20; //Decide on what to do for this case
            }
        }
    }
}