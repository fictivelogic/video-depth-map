#include "hip/hip_runtime.h"
#include <math.h>

#include "computeInitialDisparity.h"
#include "computeHalfDisparity.cu"
#include "computeFullDisparity.cu"


__global__ void

eventScheduler(int img_height,
                    int img_width,
                    const float * pixelsFullR,
                    const float * pixelsHalfR,
                    const float * pixelsQuartR,
                    const float * pixelsFullL,
                    const float * pixelsHalfL,
                    const float * pixelsQuartL,
                    float * dQuarter,
                    float * dHalf,
                    float * disparityFull,
                    const int sFull,
                    const int img_width,
                    const int img_height,
                    const bool * foregroundFullR,
                    const bool * foregroundHalfR,
                    const bool * foregroundQuarterR,
                    const bool * foregroundFullL,
                    const bool * foregroundHalfL,
                    const bool * foregroundQuarterL)
{
    dim3 blockSize(8,64); //Each thread would be responsible for 8 pixels, so a block takes care of 64 * 64 pixels
    dim3 fullGridSize( celeing(img_height/64) ,celeing(img_width/64));
    dim3 halfGridSize( celeing(img_height/(64 * 2)) ,celeing(img_width/(64*2)));
    dim3 quartGridSize( celeing(img_height/(64 * 4)) ,celeing(img_width/(64*4)));

    computeInitialDisparity  <<<quartGridSize , blockSize>>>
                        (const float * pixelsFullR,
                        const float * pixelsHalfR,
                        const float * pixelsQuartR,
                        const float * pixelsFullL,
                        const float * pixelsHalfL,
                        const float * pixelsQuartL,
                        float * dQuarter,
                        float * dHalf,
                        float * disparityFull,
                        const int sFull,
                        const int img_width,
                        const int img_height,
                        const bool * foregroundFullR,
                        const bool * foregroundHalfR,
                        const bool * foregroundQuarterR,
                        const bool * foregroundFullL,
                        const bool * foregroundHalfL,
                        const bool * foregroundQuarterL);

    computeHalfDisparity  <<<halfGridSize, blockSize>>>
                    (const float * pixelsFullR,
                     const float * pixelsHalfR,
                     const float * pixelsFullL,
                     const float * pixelsHalfL,
                     float * dQuarter,
                     float * dHalf,
                     float * disparityFull,
                     const int img_width,
                     const int img_height,
                     const bool * foregroundFullR,
                     const bool * foregroundHalfR,
                     const bool * foregroundFullL,
                     const bool * foregroundHalfL);

    computeFullDisparity <<<fullGridSize, blockSize>>>
                    (const float * pixelsFullR,
                     const float * pixelsFullL,
                     float * dHalf,
                     float * disparityFull
                     const int img_width,
                     const int img_height,
                     const bool * foregroundFullR,
                     const bool * foregroundFullL);
}