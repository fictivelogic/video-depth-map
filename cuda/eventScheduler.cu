#include "hip/hip_runtime.h"
#include <math.h>

#include "compute_disparity.cu"
// #include "computeHalfDisparity.cu"
// #include "computeFullDisparity.cu"

__global__ void eventScheduler(
    const struct pixel * imageR,
    const struct pixel * imageL,
    const int window_size,
    const int image_height,
    const int image_width,
    const bool * foregroundR,
    const bool * foregroundL,
    float * disparity_output)
{
    dim3 blockSize(8,64); //Each thread would be responsible for 8 pixels, so a block takes care of 64 * 64 pixels
    dim3 fullGridSize( celeing(image_height/64) ,celeing(image_width/64));

    computeDisparity <<<fullGridSize, blockSize>>>
    (imageR,
    imageL,
    window_size,
    image_height,
    image_width,
    foregroundR,
    foregroundL,
    disparity_output);
}