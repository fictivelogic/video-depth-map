#include "hip/hip_runtime.h"
#include <math.h>

#define IMAGE_HEIGHT 260
#define IMAGE_WIDTH 360

__global__ void

computeFullDisparity(const float * pixelsFullR,
                     const float * pixelsFullL,
                     float * dHalf,
                     float * disparityFull
                     const bool * foregroundFullR,
                     const bool * foregroundFullL)
{
    for (int i = 0; i < IMAGE_WIDTH * IMAGE_HEIGHT; i++)
    {
        if (foregroundFullL[i]==1)
        {
            float * matchingCost = new float [5];
            int k = ((i/IMAGE_WIDTH)/2) + ((i%IMAGE_WIDTH)/2);
            int prevdisp = (int) floorf(dHalf[k]);
            int initial_j = (2* prevdisp) -2;
            int max_j = 0;
            for (int j = initial_j; j <= initial_j + 4; j++)
            {
                if ( (i % IMAGE_WIDTH != 0) && ((j+i) % IMAGE_WIDTH  == 0) )
                {
                    break;
                }
                else
                {
                    if (foregroundFullR[i+j] == 1)
                    {
                        matchingCost[j - initial_j] = 0;
                        for (int m = 0; m < 3; m++)
                        {
                            matchingCost[j - initial_j] = square(pixelsQuartL[(3*i) + m] - pixelsQuartR[(3*(i+j)) +m]);
                        }
                        matchingCost[j - initial_j] /= 3;
                        max_j = j - initial_j;
                    }
                    else
                    {
                        matchingCost[j - initial_j] = 1000;
                    }
                }
            }
            float curr_min = 5000;
            for (int l = 0; l <= max_j; l++)
            {
                if (curr_min > matchingCost[l])
                {
                    curr_min = matchingCost[l];
                }
            }
            disparityFull[i] = sqrt(curr_min);
            delete [] matchingCost;
        }
        else
        {
            disparityFull[i] = 20;
        }
    }
    //copy to CPU mem
}