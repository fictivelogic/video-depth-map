#include "hip/hip_runtime.h"
__global__ void


// [260, 720, 3] 
// [260, 0-359, 3] would be LEFT [260, 360:719, 3] would be RIGHT

computeDisparity(const Color * pixelsFullR,
                 const Color * pixelsHalfR,
                 const Color * pixelsQuartR,
                 const Color * pixelsFullL,
                 const Color * pixelsHalfL,
                 const Color * pixelsQuartL,
                 const int h,
                 const int v,
                 const int sFull,
                 const bool * foregroundFullR,
                 const bool * foregroundHalfR,
                 const bool * foregroundHQuarterR,
                 const bool * foregroundFullL,
                 const bool * foregroundHalfL,
                 const bool * foregroundHQuarterL,
                 float * disparityFull)
{
    //Determine indexes based on Thread ID's

    //Initial disparity measure calculated differently
    int sQuart = sFull / 4.0;
    int dQuarter[] = new int [h * v * 0.0625]; //Assuming h and v are divisible by 4
    
    for (int i = 0; i < (h*v * 0.0625); i++)
    {
        if (foregroundHQuarterL[i] == 1)
        {
            int matchingCost[] = new int [sQuart];
            for (int j = i; j < i + sQuart; j++)
            {
                if (j+i >= (i+1)*h * 0.25) //Check if the index within windows is in pixel block row range
                {
                    break;
                }
                else
                {
                    if (foregroundHQuarterR[i+j]==1)
                    {
                        matchingCost[j-i] = 3;//compute matching cost from openCV
                    }
                    else
                    {
                        matchingCost[j-i] = 1000; //Syntax?
                    }
                }
            }
            dQuarter[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else
        {
            dQuarter[i] = 20; //Decide on what to do for this case
        }
    }

    int dHalf = new int [h*v* 0.25];
    for (int i = 0; i < (h * v * 0.25); i++)
    {
        if (foregroundHalfL[i] == 1)
        {
            int matchingCost[] = new int [4];
            int prevdisp = dQuarter[i/2]; //intentionally taking advantage of int division
            for (int j = 2 * prevdisp - 2; j < 2 * prevdisp + 2; j++)
            {
                if (j+i * h * 0.5)
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[j+i] == 1)
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 3; //compute matching cost
                    }
                    else
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 1000;
                    }
                }

            }
            dHalf[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else {
            dHalf[i] = 20;
        }
    }

    //int disparityFull = new int [h*v* 0.25];
    for (int i = 0; i < (h * v); i++)
    {
        if (foregroundFullL[i] == 1)
        {
            int matchingCost[] = new int [4];
            int prevdisp = dHalf[i/2]; //intentionally taking advantage of int division
            for (int j = 2 * prevdisp - 2; j < 2 * prevdisp + 2; j++)
            {
                if (j+i * h)
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[j+i] == 1)
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 3; //compute matching cost
                    }
                    else
                    {
                        matchingCost[j - (2 * prevdisp - 2)] = 1000;
                    }
                }

            }
            disparityFull[i] = min(matchingCost);
            delete [] matchingCost;
        }
        else {
            disparityFull[i] = 20;
        }
    }

}
