#include "hip/hip_runtime.h"
#include <math.h>
#include "computeFullDisparity.cu"

// #define IMAGE_HEIGHT 260
// #define img_width 360

__global__ void

computeHalfDisparity(const float * pixelsFullR,
                     const float * pixelsHalfR,
                     const float * pixelsFullL,
                     const float * pixelsHalfL,
                     float * dQuarter,
                     float * dHalf,
                     float * disparityFull,
                     const int img_width,
                     const int img_height,
                     const bool * foregroundFullR,
                     const bool * foregroundHalfR,
                     const bool * foregroundFullL,
                     const bool * foregroundHalfL)
{
    int ind_i = 8 * threadIdx.x + 64 * blockIdx.x;
    int ind_j = threadIdx.y + 64 * blockIdx.y;
    const int end_i = ind_i + 8;

    for (; i < end_i; i++)
    {
        if (foregroundHalfL[i]==1)
        {
            float * matchingCost = new float [5];
            int k = ((i/(img_width/2) )/2) + ((i%(img_width/2) )/2);
            int prevdisp = (int) floorf(dQuarter[k]);
            int initial_j = (2* prevdisp) -2;
            int max_j = 0;
            for (int j = initial_j; j <= initial_j + 4; j++)
            {
                if ( (i % (img_width/2) != 0) && ((j+i) % (img_width/4)  == 0) )
                {
                    break;
                }
                else
                {
                    if (foregroundHalfR[i+j] == 1)
                    {
                        matchingCost[j - initial_j] = 0;
                        for (int m = 0; m < 3; m++)
                        {
                            matchingCost[j - initial_j] = powf((pixelsQuartL[(3 * i) + m] - pixelsQuartR[(3*(i+j)) + m]),2);
                        }
                        matchingCost[j - initial_j] /= 3;
                        max_j = j - initial_j;
                    }
                    else
                    {
                        matchingCost[j - initial_j] = 1000;
                    }
                }
            }
            float curr_min = 5000;
            for (int l = 0; l <= max_j; l++)
            {
                if (curr_min > matchingCost[l])
                {
                    curr_min = matchingCost[l];
                }
            }
            dHalf[i] = sqrt(curr_min);
            delete [] matchingCost;
        }
        else
        {
            dHalf[i] = 20;
        }
    }

}