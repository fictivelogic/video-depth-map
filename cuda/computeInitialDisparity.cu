#include "hip/hip_runtime.h"
#include <math.h>
#include "computeHalfDisparity.cu"

// #define IMAGE_HEIGHT 260
// #define img_width 360

__global__ void


// [260, 720, 3] 
// [260, 0-359, 3] would be LEFT [260, 360:719, 3] would be RIGHT

computeInitialDisparity(const float * pixelsFullR,
                        const float * pixelsHalfR,
                        const float * pixelsQuartR,
                        const float * pixelsFullL,
                        const float * pixelsHalfL,
                        const float * pixelsQuartL,
                        float * dQuarter,
                        float * dHalf,
                        float * disparityFull,
                        const int sFull,
                        const int img_width,
                        const int img_height,
                        const bool * foregroundFullR,
                        const bool * foregroundHalfR,
                        const bool * foregroundQuarterR,
                        const bool * foregroundFullL,
                        const bool * foregroundHalfL,
                        const bool * foregroundQuarterL)
{

    int sQuart = sFull / 4.0; //sFull is the max window we would want for full res pixel block searching

    int ind_i = 8 * threadIdx.x + 64 * blockIdx.x;
    int ind_j = threadIdx.y + 64 * blockIdx.y;
    const int end_i = ind_i + 8;

    int i = ind_i + (img_width * ind_j);

    
    for (; i <end_i; i++)
    {
        if (foregroundQuarterL[i] == 1)
        {
            float * matchingCost = new float [sQuart];
            int max_j = 0;
            for (int j = i; j <= i + sQuart; j++)
            {
                if ( (i % (img_width/4) != 0) && ((j+i) % (img_width/4)  == 0) ) //Check if the index within windows is in pixel block row range
                {
                    break;
                }
                else
                {
                    if (foregroundQuarterR[i+j]==1)
                    {
                        matchingCost[j-i] = 0;
                        for (int m = 0; m < 3; m++)
                        {
                            matchingCost[j-i] += powf((pixelsQuartL[(3*i)+m] - pixelsQuartR[(3*(i+j))+m]), 2);
                        }
                        matchingCost[j-i] /= 3;
                        max_j = j-i;
                    }
                    else
                    {
                        matchingCost[j-i] = 1000;// might have to also update max_j
                    }
                }
            }
            float curr_min = 5000;
            for (int l = 0; l <= max_j; l++)
            {
                if (curr_min > matchingCost[l])
                {
                    curr_min = matchingCost[l];
                }
            }
            dQuarter[i] = sqrt(curr_min);
            delete [] matchingCost;
        }
        else
        {
            dQuarter[i] = 20; //Decide on what to do for this case
        }
    }
}