#include "hip/hip_runtime.h"
#include <math.h>

#define IMAGE_HEIGHT 260
#define IMAGE_WIDTH 360

__global__ void

computeFullDisparity(const float *** pixelsFullR,
                     const float *** pixelsFullL,
                     float ** dHalf,
                     const bool ** foregroundFullR,
                     const bool ** foregroundFullL)
{
    float ** disparityFull = new float * [IMAGE_HEIGHT];
    for (int k = 0; k < IMAGE_HEIGHT; k++)
    {
        disparityFull[k] = new float [IMAGE_WIDTH];
    }

    for (int k = 0; k < IMAGE_HEIGHT; k++)
    {

        for (int i = 0; i < IMAGE_WIDTH; i++)
        {
            if (foregroundFullL[k][i]==1)
            {
                float * matchingCost = new float [5];
                int prevdisp = (int) floorf(dHalf[k/2][i/2]);
                int initial_j = (2* prevdisp) -2;
                int max_j = 0;
                for (int j = initial_j; j <= initial_j + 4; j++)
                {
                    if ( j+i >= IMAGE_WIDTH)
                    {
                        break;
                    }
                    else
                    {
                        if (foregroundFullR[i+j] == 1)
                        {
                            matchingCost[j - initial_j] = 0;
                            for (int m = 0; m < 3; m++)
                            {
                                matchingCost[j - initial_j] = square(pixelsQuartL[k][i][m] - pixelsQuartR[k][i+j][m]);
                            }
                            matchingCost[j - initial_j] /= 3;
                            max_j = j - initial_j;
                        }
                        else
                        {
                            matchingCost[j - initial_j] = 1000;
                        }
                    }
                }
                float curr_min = 5000;
                for (int l = 0; l <= max_j; l++)
                {
                    if (curr_min > matchingCost[l])
                    {
                        curr_min = matchingCost[l];
                    }
                }
                disparityFull[k][i] = sqrt(curr_min);
                delete [] matchingCost;
            }
            else
            {
                disparityFull[k][i] = 20;
            }
        }
    }
    for (int i = 0; i < IMAGE_HEIGHT/2; i++)
    {
        delete [] dHalf[i];
    }
    delete [] dHalf;
}