#include "hip/hip_runtime.h"
#include <math.h>
#include "computeHalfDisparity.cu"

#define IMAGE_HEIGHT 260
#define IMAGE_WIDTH 360

__global__ void


// [260, 720, 3] 
// [260, 0-359, 3] would be LEFT [260, 360:719, 3] would be RIGHT

computeInitialDisparity(const float *** pixelsFullR,
                        const float *** pixelsHalfR,
                        const float *** pixelsQuartR,
                        const float *** pixelsFullL,
                        const float *** pixelsHalfL,
                        const float *** pixelsQuartL,
                        const int sFull,
                        const bool ** foregroundFullR,
                        const bool ** foregroundHalfR,
                        const bool ** foregroundQuarterR,
                        const bool ** foregroundFullL,
                        const bool ** foregroundHalfL,
                        const bool ** foregroundQuarterL)
{
    int sQuart = sFull / 4.0; //sFull is the max window we would want for full res pixel block searching
    float ** dQuarter = new float * [IMAGE_HEIGHT/4];
    for (int k = 0; k < IMAGE_HEIGHT/4; k++)
    {
        dQuarter[k] = new float [IMAGE_WIDTH/4];
    }
    
    for (int k = 0; k < IMAGE_HEIGHT/4; k++)
    {
        for (int i = 0; i < IMAGE_WIDTH/4; i++)
        {
            if (foregroundQuarterL[k][i] == 1)
            {
                float * matchingCost = new float [sQuart];
                int max_j = 0;
                for (int j = i; j <= i + sQuart; j++)
                {
                    if (j+i >= IMAGE_WIDTH/4) //Check if the index within windows is in pixel block row range
                    {
                        break;
                    }
                    else
                    {
                        if (foregroundQuarterR[k][i+j]==1)
                        {
                            matchingCost[j-i] = 0;
                            for (int m = 0; m < 3; m++)
                            {
                                matchingCost[j-i] += square(pixelsQuartL[k][i][m] - pixelsQuartR[k][i+j][m]);
                            }
                            matchingCost[j-i] /= 3;
                            max_j = j;
                        }
                        else
                        {
                            matchingCost[j-i] = 1000;// might have to also update max_j
                        }
                    }
                }
                float curr_min = 5000;
                for (int l = 0; l <= max_j; l++)
                {
                    if (curr_min > matchingCost[l])
                    {
                        curr_min = matchingCost[l];
                    }
                }
                dQuarter[k][i] = sqrt(curr_min);
                delete [] matchingCost;
            }
            else
            {
                dQuarter[k][i] = 20; //Decide on what to do for this case
            }
        }
    }
    
    computeHalfDisparity(const float *** pixelsFullR,
                     const float *** pixelsHalfR,
                     const float *** pixelsFullL,
                     const float *** pixelsHalfL,
                     float ** dQuarter,
                     const bool ** foregroundFullR,
                     const bool ** foregroundHalfR,
                     const bool ** foregroundFullL,
                     const bool ** foregroundHalfL);

    for (int i = 0; i < IMAGE_HEIGHT/4; i++)
    {
        delete [] dQuarter[i];
    }
    delete [] dQuarter;
}