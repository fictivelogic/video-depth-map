#include "hip/hip_runtime.h"
__global__ void


computeDisparity(const Color * pixelsFullR,
                 const Color * pixelsHalfR,
                 const Color * pixelsQuartR,
                 const Color * pixelsFullL,
                 const Color * pixelsHalfL,
                 const Color * pixelsQuartL,
                 const int h,
                 const int v,
                 const int sFull,
                 const bool * foregroundFullR,
                 const bool * foregroundHalfR,
                 const bool * foregroundHQuarterR,
                 const bool * foregroundFullL,
                 const bool * foregroundHalfL,
                 const bool * foregroundHQuarterL,
                 float * disparityFull)
{
    //Determine indexes based on Thread ID's

    //Initial disparity measure calculated differently
    int sQuart = sFull / 4;
    float dQuarter[] = new float [h * v * 0.0625]; //Assuming h and v are divisible by 4
    
    for (int i = 0; i < h*v; i++)
    {
        if foregroundHQuarterR[i] == 1
        {
            for (int j = 0; j < sQuart; j++)
            {
                if (j+i >= (i+1)*h)
                    break
                else
                {
                    if (foregroundHQuarterL[i+j]==1)
                    {
                        matchingCost[j] = //compute matching cost from openCV
                    }
                    else
                    {
                        matchingCost[j] = inf; //Syntax?
                    }
                }
            }
            dQuarter[i] = min(matchingCost);
        }
        else
            dQuarter[i] = 20; //Decide on what to do for this case
    }

}

createFinalHiddenFeaturesKernel(const float *weights,
    const float *movie_rating_probs, float* final_hidden_feature_probs,
    int num_movies, int num_hidden_features) {

    // weights[NUM_MOVIES][5][NUM_FEATURES]
    // movie_rating_probs[NUM_MOVIES][5]
    // final_hidden_feature_probs[NUM_FEATURES]
    unsigned int hidden_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int movie_id = 0;
    unsigned int rating = 0;
    float dot_prod; // Temporary, local dot product variable
    while (hidden_id < num_hidden_features) {
        dot_prod = 0.00; // Initialize the dot product to 0

        for (movie_id = 0; movie_id < num_movies; movie_id++) {
            for (rating = 0; rating < 5; rating++) {
                // Indexing: weights[movie_id][rating][feature_id]
                // movie_id - [1, 17771]
                // rating - [0, 4]
                // hidden_id - [0, 99]
                // Do the dot product
                dot_prod += weights[movie_id*5*num_hidden_features
                                    + rating*num_hidden_features
                                    + hidden_id]
                            * final_hidden_feature_probs[hidden_id];
            }
        }
        // Store the dot_product result
        final_hidden_feature_probs[hidden_id] = dot_prod;

        // Re-use this thread on another data point:
        hidden_id += blockDim.x * gridDim.x;
    }
}
