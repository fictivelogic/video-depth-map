
#include <hip/hip_runtime.h>
// Simplified attempt
#define STARTING_MATCHING_COST 100.0f
struct pixel {
    float R;
    float G;
    float B;
};

__global__ void computeDisparity(
    const struct pixel * imageR,  // Input pixel array of left image 
                           // array dim: image[image_width][image_height]
    const struct pixel * imageL,  // Input pixel array of right image
    const int window_size, // size of window used for block searching
    const int image_height, // height of the image, in pixels
    const int image_width, // width of the image, in pixels
    const bool * foregroundR, // Array of foreground flags for each pixel in imageR
    const bool * foregroundL, // Array of foreground flags for each pixel in imageL
    float * disparity_output) // Output array (same dim as imageR/L) of disparity values
{
    // Local variables:
    long int pixel_index = blockIdx.x * blockDim.x + threadIdx.x; // Index of current pixel
    // TODO: Assign proper indexing   
    float matching_cost = 0.0;
    float min_matching_cost = STARTING_MATCHING_COST;
    long int min_cost_offset = 0; 
    long int offset_pixel_index = 0;

    while(pixel_index < image_height * image_width) { 
        // while... the thread index hasn't gone outside the image dimensions
        if (foregroundL[pixel_index] == 1) {
            // Calculate matching cost for this foreground pixel
            // ensure that we are not going over the end of the pixel row
            offset_pixel_index = pixel_index;
            min_matching_cost = STARTING_MATCHING_COST;
            min_cost_offset = 0;
            for (int offset = 0; offset < window_size; offset++) { 
                if ((pixel_index % image_width) + offset >= image_width) {
                    break;
                }
                matching_cost =  powf(imageL[pixel_index].R - imageR[offset_pixel_index].R, 2);
                matching_cost += powf(imageL[pixel_index].G - imageR[offset_pixel_index].G, 2);
                matching_cost += powf(imageL[pixel_index].B - imageR[offset_pixel_index].B, 2);
                if (matching_cost < min_matching_cost) {
                    min_matching_cost = matching_cost; 
                    min_cost_offset = offset;
                }
                offset_pixel_index++;
            }

            disparity_output[pixel_index] = min_cost_offset; // TODO: What exactly IS disparity?
        } else {
            disparity_output[pixel_index] = -1;
        }
        pixel_index += blockDim.x * gridDim.x;
    }
}

