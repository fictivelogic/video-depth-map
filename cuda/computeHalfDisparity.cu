#include "hip/hip_runtime.h"
#include <math.h>
#include "computeFullDisparity.cu"

#define IMAGE_HEIGHT 260
#define IMAGE_WIDTH 360

__global__ void

computeHalfDisparity(const float *** pixelsFullR,
                     const float *** pixelsHalfR,
                     const float *** pixelsFullL,
                     const float *** pixelsHalfL,
                     float ** dQuarter,
                     const bool ** foregroundFullR,
                     const bool ** foregroundHalfR,
                     const bool ** foregroundFullL,
                     const bool ** foregroundHalfL)
{
    float ** dHalf = new float * [IMAGE_HEIGHT/2];
    for (int k = 0; k < IMAGE_HEIGHT/2; k++)
    {
        dHalf[k] = new float [IMAGE_WIDTH/2];
    }

    for (int k = 0; k < IMAGE_HEIGHT/2; k++)
    {

        for (int i = 0; i < IMAGE_WIDTH/2; i++)
        {
            if (foregroundHalfL[k][i]==1)
            {
                float * matchingCost = new float [5];
                int prevdisp = (int) floorf(dQuarter[k/2][i/2]);
                int initial_j = (2* prevdisp) -2;
                int max_j = 0;
                for (int j = initial_j; j <= initial_j + 4; j++)
                {
                    if ( j+i >= IMAGE_WIDTH/4)
                    {
                        break;
                    }
                    else
                    {
                        if (foregroundHalfR[i+j] == 1)
                        {
                            matchingCost[j - initial_j] = 0;
                            for (int m = 0; m < 3; m++)
                            {
                                matchingCost[j - initial_j] = square(pixelsQuartL[k][i][m] - pixelsQuartR[k][i+j][m]);
                            }
                            matchingCost[j - initial_j] /= 3;
                            max_j = j - initial_j;
                        }
                        else
                        {
                            matchingCost[j - initial_j] = 1000;
                        }
                    }
                }
                float curr_min = 5000;
                for (int l = 0; l <= max_j; l++)
                {
                    if (curr_min > matchingCost[l])
                    {
                        curr_min = matchingCost[l];
                    }
                }
                dHalf[k][i] = sqrt(curr_min);
                delete [] matchingCost;
            }
            else
            {
                dHalf[k][i] = 20;
            }
        }
    }
    computeFullDisparity(const float *** pixelsFullR,
                     const float *** pixelsFullL,
                     float ** dHalf,
                     const bool ** foregroundFullR,
                     const bool ** foregroundFullL);

    for (int i = 0; i < IMAGE_HEIGHT; i++)
    {
        delete [] disparityFull[i];
    }
    delete [] disparityFull;
}